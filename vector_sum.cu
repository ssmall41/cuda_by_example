
#include <hip/hip_runtime.h>
#include <stdio.h>

// Adds two vectors. The maximum number of blocks is a limitation.
__global__ void sum_vectors(int* a, int* b, int* c, int size)
{
	int i = blockIdx.x;
	if(i < size)
		c[i] = a[i] + b[i];
}

// Adds two vectors using threads and blocks.
__global__ void sum_vectors_arbitrary(int* a, int* b, int* c, int size)
{
	int i;
	for(i=threadIdx.x+blockIdx.x*blockDim.x; i<size; i+=blockDim.x*gridDim.x)
		c[i] = a[i] + b[i];
}

int main(int argc, char* argv[])
{
	int size = 20, i;
	int* a = (int*) malloc(size*sizeof(int));
	int* b = (int*) malloc(size*sizeof(int));
	int* c = (int*) malloc(size*sizeof(int));
	int *dev_a, *dev_b, *dev_c;
	for(i=0;i<size;i++)
	{
		a[i] = i;
		b[i] = 2*i;
	}

	hipMalloc(&dev_a, size*sizeof(int));
	hipMalloc(&dev_b, size*sizeof(int));
	hipMalloc(&dev_c, size*sizeof(int));
	
	hipMemcpy(dev_a, a, size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size*sizeof(int), hipMemcpyHostToDevice);
	
	//sum_vectors<<<size, 1>>>(dev_a, dev_b, dev_c, size);
	sum_vectors_arbitrary<<<2, 2>>>(dev_a, dev_b, dev_c, size);

	hipMemcpy(c, dev_c, size*sizeof(int), hipMemcpyDeviceToHost);
	
	for(i=0;i<size;i++)
		printf("%i ", c[i]);
	printf("\n");
	
	hipFree(dev_c);
	hipFree(dev_b);
	hipFree(dev_a);
	free(c);
	free(b);
	free(a);
	
	return 0;
}

