//Complie: nvcc info.cu
//Run: ./a.out

#include <stdio.h>

int main()
{
    hipDeviceProp_t  prop;

    int count,i;
    hipError_t val = hipGetDeviceCount( &count );
    if(val != hipSuccess)
    {
	printf("Error getting device count:\n");
	if(val == hipErrorNoDevice)
		printf("No device\n");
	else if(val == hipErrorInsufficientDriver)
		printf("Insufficient driver\n");
	else
	{
		printf("Got error %i\n", val);
		const char* errorName = hipGetErrorName(val);
		const char* errorString = hipGetErrorString(val);
		printf("Error name: %s.\n", errorName);
		printf("Error string: %s.\n", errorString);
	}
	return 1;
    }

    for (i=0; i<count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        
        /* Deprecated for asyncEngineCount
        printf( "Device copy overlap:  " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n");
        */
        
        printf("Number of async engines: %i\n", prop.asyncEngineCount);
            
        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );
        printf("Concurrent kernel executions: %i\n", prop.concurrentKernels);

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n",
                    prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",
                    prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                    prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                    prop.maxGridSize[0], prop.maxGridSize[1],
                    prop.maxGridSize[2] );
        printf( "\n" );
	printf("***********\n");
    }

	return 0;
}

