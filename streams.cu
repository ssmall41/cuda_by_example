// Tests out streaming.
// One stream needs ~370ms, while 2 need ~290ms.
// There's not much difference in depth vs breadth first. However,
// for larger data sizes, depth seems to be slightly better.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

/* This is taken directly from the CUDA by Example book, chapter 10.
 * It doesn't really do anything useful, just represents some work
 * on the device. */
__global__ void kernel(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < N)
	{
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0;
		c[idx] = (as + bs) / 2;
	}
}


int main(int argc, char* argv[])
{
	int i;
	float elapsed_time;

	// Create events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Create streams
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	//stream1 = stream0;

	// Initialize memory
	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;
	hipMalloc(&dev_a0, N*sizeof(int));
	hipMalloc(&dev_b0, N*sizeof(int));
	hipMalloc(&dev_c0, N*sizeof(int));
	hipMalloc(&dev_a1, N*sizeof(int));
	hipMalloc(&dev_b1, N*sizeof(int));
	hipMalloc(&dev_c1, N*sizeof(int));
	hipHostAlloc(&host_a, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_b, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&host_c, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);

	for(i=0;i<FULL_DATA_SIZE;i++)
	{
		host_a[i] = i;
		host_b[i] = i;
	}

	hipEventRecord(start, 0);

	for(int i=0;i<FULL_DATA_SIZE;i+=N*2)
	{


		// Depth first, incorrect
		// Stream 0
		hipMemcpyAsync(dev_a0, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b0, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
		kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
		hipMemcpyAsync(host_c+i, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0);

		// Stream 1
		hipMemcpyAsync(dev_a1, host_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b1, host_b+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1);
		kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
		hipMemcpyAsync(host_c+i+N, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1);

/*
		// Breadth first, correct
		cudaMemcpyAsync(dev_a0, host_a+i, N*sizeof(int), cudaMemcpyHostToDevice, stream0);
		cudaMemcpyAsync(dev_a1, host_a+i+N, N*sizeof(int), cudaMemcpyHostToDevice, stream1);

		cudaMemcpyAsync(dev_b0, host_b+i, N*sizeof(int), cudaMemcpyHostToDevice, stream0);
		cudaMemcpyAsync(dev_b1, host_b+i+N, N*sizeof(int), cudaMemcpyHostToDevice, stream1);

		kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
		kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

		cudaMemcpyAsync(host_c+i, dev_c0, N*sizeof(int), cudaMemcpyDeviceToHost, stream0);
		cudaMemcpyAsync(host_c+i+N, dev_c1, N*sizeof(int), cudaMemcpyDeviceToHost, stream1);
*/
	}

	// Sync
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	// Check timing
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("Total time taken is %3.1f ms.\n", elapsed_time);

	// Trash memory
	hipHostFree(host_c);
	hipHostFree(host_b);
	hipHostFree(host_a);
	hipFree(dev_c1);
	hipFree(dev_b1);
	hipFree(dev_a1);
	hipFree(dev_c0);
	hipFree(dev_b0);
	hipFree(dev_a0);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream0);
	hipEventDestroy(stop);
	hipEventDestroy(start);

	return 0;
}
