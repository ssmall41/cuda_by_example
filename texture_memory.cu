// Tries texture memory. Can be compared against untexture_memory.cu.
// Using texture memory seems to require MORE time here.
// It seems using dim as a multiple of prop.texturePitchAlignment helps.
// I suspect the resource descriptor or so was suppose to use this somehow.


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void copy_heater_buffer(hipTextureObject_t tex_heater_buffer, float* in_buffer, int dim)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	
	float heater_value = tex1Dfetch<float>(tex_heater_buffer, offset);
	if(x < dim && y < dim && heater_value > 0.0)
		in_buffer[offset] = heater_value;
}

__global__ void heat_spread(float* out_buffer, hipTextureObject_t tex_in_buffer, int dim, float speed)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	
	if(x > dim || y > dim)	return;
	
	int l = offset - 1;
	int r = offset + 1;
	if(x == 0) l++;
	if(x == dim-1) r--;
	
	int top = offset - dim;
	int bot = offset + dim;
	if(y == 0) top += dim;
	if(y == dim-1) bot -= dim;
	
	float sum_all_directions = tex1Dfetch<float>(tex_in_buffer, l) + tex1Dfetch<float>(tex_in_buffer, r) + tex1Dfetch<float>(tex_in_buffer, top) + tex1Dfetch<float>(tex_in_buffer, bot);
	out_buffer[offset] = tex1Dfetch<float>(tex_in_buffer, offset) + speed * (sum_all_directions - tex1Dfetch<float>(tex_in_buffer, offset)*4);
}

void swap_tex_buffers(hipTextureObject_t* tex_in_buffer, hipTextureObject_t* tex_out_buffer)
{
	hipTextureObject_t holder = *tex_in_buffer;
	*tex_in_buffer = *tex_out_buffer;
	*tex_out_buffer = holder;
}

void print_buffer(float* buffer, int size)
{
	int i, j;
	printf("#########\n");
	for(i=0;i<size;i++)
	{
		for(j=0;j<size;j++)
			printf("%f ", buffer[i + j*size]);
		printf("\n");
	}
}


void print_device_buffer(float* dev_buffer, int size)
{
	float* temp_buffer = (float*) malloc(size*size*sizeof(float));
	hipMemcpy(temp_buffer, dev_buffer, size*size*sizeof(float), hipMemcpyDeviceToHost);
	print_buffer(temp_buffer, size);
	free(temp_buffer);
}

hipTextureObject_t define_texture_memory(float* dev_data, int size)
{
	// Define the Resource Descriptor
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	//struct cudaResourceDesc* resDesc = (struct cudaResourceDesc*) calloc(1, sizeof(struct cudaResourceDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = dev_data;
	resDesc.res.linear.sizeInBytes = size*size*sizeof(float);
	resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
	resDesc.res.linear.desc.x = 32;

	// Define the Texture Descriptor
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	//struct cudaTextureDesc* texDesc = (struct cudaTextureDesc*) calloc(1, sizeof(struct cudaTextureDesc));
	texDesc.readMode = hipReadModeElementType;

	// Create the Texture Object
	hipTextureObject_t tex;
	//cudaCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
	
	return tex;
}

float* get_devPtr(hipTextureObject_t texObject)
{
	hipResourceDesc desc;
	hipGetTextureObjectResourceDesc (&desc, texObject);
	return (float*) desc.res.linear.devPtr;
}

int main(int argc, char* argv[])
{
	int i, dim = 32, num_iterations = 20000;
	dim3 num_blocks(2, 2), num_threads(dim/2, dim/2);
	//dim3 num_blocks(1, 1), num_threads(dim, dim);
	float speed = 0.2;
	int buffer_size = dim*dim;
	
	// Create events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Reserve memory
	//cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc(32, 0, 0, 0, cudaChannelFormatKindFloat);
	float* heater_buffer = (float*) calloc(buffer_size, sizeof(float));
	float *dev_heater_buffer, *dev_in_buffer, *dev_out_buffer;
	hipMalloc(&dev_heater_buffer, buffer_size*sizeof(float));
	hipMalloc(&dev_in_buffer, buffer_size*sizeof(float));
	hipMalloc(&dev_out_buffer, buffer_size*sizeof(float));
	hipTextureObject_t tex_heater_buffer = define_texture_memory(dev_heater_buffer, dim);
	hipTextureObject_t tex_in_buffer = define_texture_memory(dev_in_buffer, dim);
	hipTextureObject_t tex_out_buffer = define_texture_memory(dev_out_buffer, dim);
	
	// Initialize the heaters
	heater_buffer[0 + 0*dim] = 1.0;
	heater_buffer[1 + 2*dim] = 1.0;
	//heater_buffer[2 + 5*dim] = 0.5;
	//heater_buffer[6 + 7*dim] = 1.0;
	
	// Send data to the GPU
	hipMemcpy(dev_heater_buffer, heater_buffer, buffer_size*sizeof(float), hipMemcpyHostToDevice);

	//print_device_buffer(dev_heater_buffer, dim);

	// Let the heat spread
	hipEventRecord(start, 0);
	for(i=0;i<num_iterations;i++)
	{
		//printf("Iteration %i\n", i);
		copy_heater_buffer<<<num_blocks, num_threads>>>(tex_heater_buffer, dev_in_buffer, dim);
		//print_device_buffer(dev_in_buffer, dim);
		heat_spread<<<num_blocks, num_threads>>>(dev_out_buffer, tex_in_buffer, dim, speed);
		//print_device_buffer(dev_out_buffer, dim);

		swap_tex_buffers(&tex_in_buffer, &tex_out_buffer);
		dev_in_buffer = get_devPtr(tex_in_buffer);  // Needed in order to switch the dev pointers
		dev_out_buffer = get_devPtr(tex_out_buffer);
	}
	copy_heater_buffer<<<num_blocks, num_threads>>>(tex_heater_buffer, dev_in_buffer, dim);
	hipEventRecord(stop, 0);

	// Copy the input buffer to the host
	hipMemcpy(heater_buffer, dev_in_buffer, buffer_size*sizeof(float), hipMemcpyDeviceToHost);
	
	// Print for sanity
	//print_buffer(heater_buffer, dim);

	//Check the runtime
	float elapsed_time;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("Total time %3.2f ms\n", elapsed_time);

	// Free memory
	hipDestroyTextureObject(tex_out_buffer);
	hipDestroyTextureObject(tex_in_buffer);
	hipDestroyTextureObject(tex_heater_buffer);
	hipFree(dev_out_buffer);
	hipFree(dev_in_buffer);
	hipFree(dev_heater_buffer);
	free(heater_buffer);

	return 0;
}

