// Tests using an atomic operation to see how it can all go wrong.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void lots_o_incs(int* value)
{
	// Safe
	atomicAdd(value, 1);

	// Not safe
	//(*value) += 1;
}


int main(int argc, char* argv[])
{
	int *dev_value, value = 0;
	hipMalloc(&dev_value, 1*sizeof(int));
	hipMemcpy(dev_value, &value, 1*sizeof(int), hipMemcpyHostToDevice);

	lots_o_incs<<<2, 32>>>(dev_value);

	hipMemcpy(&value, dev_value, 1*sizeof(int), hipMemcpyDeviceToHost);

	printf("Value is %i\n", value);

	hipFree(dev_value);

	return 0;
}
